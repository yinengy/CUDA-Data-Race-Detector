/* 
 *
 * A NVBit tool, which will detect conflict memory access in the kernel.
 * The raw output will be processed by a Pytyhon script
 *
 * Yineng Yan (yinengy@umich.edu), 2020
 */

#include <stdio.h>
#include <unordered_set>

/* header for every nvbit tool */
#include "nvbit_tool.h"

/* interface of nvbit */
#include "nvbit.h"

/* nvbit utility functions */
#include "utils/utils.h"

/* Set used to avoid re-instrumenting the same functions multiple times */
std::unordered_set<hipFunction_t> already_instrumented;

/* instrument functions, it follows the code of the sample tools in NVbit release */
void instrument_function_if_needed(hipCtx_t ctx, hipFunction_t func) {
    /* Get related functions of the kernel (device function that can be
     * called by the kernel) */
    std::vector<hipFunction_t> related_functions =
        nvbit_get_related_functions(ctx, func);

    /* add kernel itself to the related function vector */
    related_functions.push_back(func);

    /* iterate on function */
    for (auto f : related_functions) {
        /* "recording" function was instrumented, if set insertion failed
         * we have already encountered this function */
        if (!already_instrumented.insert(f).second) {
            continue;
        }

        const std::vector<Instr *> &instrs = nvbit_get_instrs(ctx, f);
        for (auto instr : instrs) {
            // only instrument load & store instructions
            if (instr->getMemOpType()!=Instr::memOpType::GLOBAL
                    && instr->getMemOpType()!=Instr::memOpType::SHARED) {
                continue;
            }

            for (int i = 0; i < instr->getNumOperands(); i++) {
                /* get the operand "i" */
                const Instr::operand_t *op = instr->getOperand(i);

                if (op->type == Instr::operandType::MREF) {
                    /* insert call to the instrumentation function with its
                     * arguments */
                    nvbit_insert_call(instr, "print_ldst", IPOINT_AFTER);
                    /* predicate value */
                    nvbit_add_call_arg_pred_val(instr);
                    /* memory reference 64 bit address */
                    nvbit_add_call_arg_mref_addr64(instr);
                    nvbit_add_call_arg_const_val32(instr, instr->getMemOpType()==Instr::memOpType::SHARED);
                    nvbit_add_call_arg_const_val32(instr, instr->isLoad());
                    break;
                }
            }
        }
    }
}

/* This call-back is triggered every time a CUDA driver call is encountered.
 * Here we can look for a particular CUDA driver call by checking at the
 * call back ids  which are defined in tools_cuda_api_meta.h.
 * This call back is triggered bith at entry and at exit of each CUDA driver
 * call, is_exit=0 is entry, is_exit=1 is exit.
 * */
void nvbit_at_cuda_event(hipCtx_t ctx, int is_exit, nvbit_api_cuda_t cbid,
                         const char *name, void *params, hipError_t *pStatus) {
    /* Identify all the possible CUDA launch events */
    if (cbid == API_CUDA_cuLaunch || cbid == API_CUDA_cuLaunchKernel_ptsz ||
        cbid == API_CUDA_cuLaunchGrid || cbid == API_CUDA_cuLaunchGridAsync ||
        cbid == API_CUDA_cuLaunchKernel) {
        /* cast params to cuLaunch_params since if we are here we know these are
         * the right parameters type */
        cuLaunch_params *p = (cuLaunch_params *)params;

        if (!is_exit) {
            instrument_function_if_needed(ctx, p->f);
            nvbit_enable_instrumented(ctx, p->f, true);
        }
    }
}
